#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include <math.h>

#define PI 3.14159265358979323846

__global__
void function(size_t N, double* xr, double* xi, double* x)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;


    double theta;

    for (int k = index; k < N; k += stride) {
        xr[k] = 0;
        xi[k] = 0;
        for (int n = 0; n < N; n++) {
            theta = (2 * PI * k * n) / N;
            xr[k] = xr[k] + x[n] * cos(theta);
            xi[k] = xi[k] - x[n] * sin(theta);
        }
    }

}
__global__
void function2(size_t N, double* xr, double* xi, double* y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    double theta;
    for (int n = index; n < N; n += stride) {
        y[n] = 0;
        for (int k = 0; k < N; k++) {
            theta = (2 * PI * k * n) / (double)N;
            y[n] = y[n] + xr[k] * cos(theta) - xi[k] * sin(theta);
        }
        y[n] = y[n] / (double)N;

    }

}

int main() {
    const size_t ARRAY_SIZE = 5;//1<<8;
    //const size_t ARRAY_SIZE = 1<<10;
    //const size_t ARRAY_SIZE = 1<<24;
    //const size_t ARRAY_SIZE = 1<<26;
    //const size_t ARRAY_SIZE = 1<<28;
    const size_t ARRAY_BYTES = ARRAY_SIZE * sizeof(double);
    //number of times the program is to be executed
    const size_t loope = 1;
    //declare array

    int device = -1;
    hipGetDevice(&device);
    double* xr, * xi, * x, * y;
    hipMallocManaged(&xr, ARRAY_BYTES);
    hipMallocManaged(&xi, ARRAY_BYTES);
    hipMallocManaged(&x, ARRAY_BYTES);
    hipMallocManaged(&y, ARRAY_BYTES);
    //mem advise
    hipMemAdvise(x, ARRAY_BYTES, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(x, ARRAY_BYTES, hipMemAdviseSetReadMostly, hipCpuDeviceId);
    //page creation
    hipMemPrefetchAsync(x, ARRAY_BYTES, hipCpuDeviceId, NULL);
    hipMemPrefetchAsync(xr, ARRAY_BYTES, device, NULL);
    hipMemPrefetchAsync(xi, ARRAY_BYTES, device, NULL);
    hipMemPrefetchAsync(y, ARRAY_BYTES, device, NULL);
    // init array
    for (int i = 0;i < ARRAY_SIZE;i++) {
        x[i] = (double)i;
    }

    for (size_t i = 0; i < ARRAY_SIZE;i++) {
        printf("y[%d] = %.2f\n", i, x[i]);
    }
    //prefetch
    hipMemPrefetchAsync(x, ARRAY_BYTES, device, NULL);

    // setup CUDA kernel
    size_t numThreads = 256;
    //size_t numThreads = 512;
    //size_t numThreads = 1024;
    //size_t numBlocks = 1;
    size_t numBlocks = (ARRAY_SIZE + numThreads - 1) / numThreads;
    printf("*** function ***\n");
    printf("numElements = %lu\n", ARRAY_SIZE);
    printf("numBlocks = %lu, numThreads = %lu \n", numBlocks, numThreads);
    for (size_t i = 0; i < loope;i++) {
        function << <numBlocks, numThreads >> > (ARRAY_SIZE, xr, xi, x);
    }

    //barrier
    hipDeviceSynchronize();
    hipMemPrefetchAsync(x, ARRAY_BYTES, hipCpuDeviceId, NULL);
    hipMemPrefetchAsync(xr, ARRAY_BYTES, hipCpuDeviceId, NULL);
    hipMemPrefetchAsync(xi, ARRAY_BYTES, hipCpuDeviceId, NULL);
    //error checking
    /*for (size_t i = 0; i < ARRAY_SIZE;i++) {
        printf("%.3f + j(%.5f)\n", xr[i], xi[i]);
    }*/


    //mem advise
    hipMemAdvise(xr, ARRAY_BYTES, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(xr, ARRAY_BYTES, hipMemAdviseSetReadMostly, hipCpuDeviceId);
    hipMemAdvise(xi, ARRAY_BYTES, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(xi, ARRAY_BYTES, hipMemAdviseSetReadMostly, hipCpuDeviceId);
    //page creation
    

    hipMemPrefetchAsync(xr, ARRAY_BYTES, device, NULL);
    hipMemPrefetchAsync(xi, ARRAY_BYTES, device, NULL);
    for (size_t i = 0; i < loope;i++) {
        function2 << <numBlocks, numThreads >> > (ARRAY_SIZE, xr, xi, y);
    }

    ////barrier
    hipDeviceSynchronize();
    hipMemPrefetchAsync(y, ARRAY_BYTES, hipCpuDeviceId, NULL);
    /*for (size_t i = 0; i < ARRAY_SIZE;i++) {
        printf("y[%d] = %.2f\n", i, x[i]);
    }*/
    size_t err_count = 0;
    for (int i = 0; i < ARRAY_SIZE; i++) {
        if (x[i] != y[i]) {
            err_count++;
        }
    }
    printf("Error count(CUDA program): %zu\n", err_count);
    //free memory
    hipFree(xr);
    hipFree(xi);
    hipFree(x);
    return 0;
}
/*#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}*/
